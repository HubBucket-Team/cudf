/*
 * Copyright 2019 BlazingDB, Inc.
 *     Copyright 2019 Alexander Ocsa <alexander@blazingdb.com>
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "groupby_wo_valid.h"
#include "rmm/thrust_rmm_allocator.h"

#include <hip/hip_runtime.h>

#include "copying.hpp"
#include "cudf.h"
#include "utilities/error_utils.hpp"

#include "groupby/aggregation_operations.hpp"

#include "rmm/rmm.h"
#include "utilities/cudf_utils.h"

#include <limits>
#include <memory>
#include <table/device_table.cuh>
#include <table/device_table_row_operators.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/gather.h>

namespace {


template <typename aggregation_type, typename aggregation_operation>
gdf_error GroupbySort(gdf_size_type num_groupby_cols, int32_t *d_sorted_indices,
                      gdf_column *in_groupby_columns[],
                      const aggregation_type *const in_aggregation_column,
                      gdf_column *out_groupby_columns[],
                      aggregation_type *out_aggregation_column,
                      gdf_size_type *out_size,
                      aggregation_operation aggregation_op, gdf_context *ctxt) {
  int32_t nrows = in_groupby_columns[0]->size;

  auto device_input_table =
      device_table::create(num_groupby_cols, &(in_groupby_columns[0]));
  auto comp = row_equality_comparator<false>(*device_input_table, true);

  hipStream_t stream;
  hipStreamCreate(&stream);
  auto exec = rmm::exec_policy(stream)->on(stream);

  auto agg_col_iter = thrust::make_permutation_iterator(in_aggregation_column,
                                                        d_sorted_indices);

  auto ret = thrust::reduce_by_key(
      exec, d_sorted_indices, d_sorted_indices + nrows, agg_col_iter,
      d_sorted_indices, out_aggregation_column, comp, aggregation_op);
  gdf_size_type new_size = thrust::distance(out_aggregation_column, ret.second);
  *out_size = new_size;

  // run gather operation to establish new order
  cudf::table table_in(in_groupby_columns, num_groupby_cols);
  cudf::table table_out(out_groupby_columns, num_groupby_cols);

  cudf::gather(&table_in, d_sorted_indices, &table_out);

  for (int i = 0; i < num_groupby_cols; i++) {
    out_groupby_columns[i]->size = new_size;
  }

  return GDF_SUCCESS;
}

template <typename aggregation_type, template <typename T> class op>
gdf_error typed_groupby_sort(gdf_size_type num_groupby_cols,
                             gdf_column *in_groupby_columns[],
                             gdf_column *in_aggregation_column,
                             gdf_column *out_groupby_columns[],
                             gdf_column *out_aggregation_column,
                             gdf_context *ctxt,
                             rmm::device_vector<int32_t> &sorted_indices) {
  using op_type = op<aggregation_type>;
  // Cast the void* data to the appropriate type
  aggregation_type *in_agg_col =
      static_cast<aggregation_type *>(in_aggregation_column->data);
  aggregation_type *out_agg_col =
      static_cast<aggregation_type *>(out_aggregation_column->data);
  gdf_size_type output_size{0};
  gdf_error gdf_error_code = GDF_SUCCESS;

  gdf_error_code =
      GroupbySort(num_groupby_cols, sorted_indices.data().get(),
                  in_groupby_columns, in_agg_col, out_groupby_columns,
                  out_agg_col, &output_size, op_type(), ctxt);
  out_aggregation_column->size = output_size;
  return gdf_error_code;
}

template <template <typename T> class op> struct dispatch_groupby_forwarder {
  template <typename TypeAgg, typename... Ts>
  gdf_error operator()(Ts &&... args) {
    return typed_groupby_sort<TypeAgg, op>(std::forward<Ts>(args)...);
  }
};

} // namespace

gdf_error gdf_group_by_sort_wo_valids(
    gdf_size_type ncols, gdf_column *in_groupby_columns[],
    gdf_column *in_aggregation_column, gdf_column *out_groupby_columns[],
    gdf_column *out_aggregation_column, gdf_agg_op agg_op, gdf_context *ctxt,
    rmm::device_vector<int32_t> &sorted_indices) {

  // Make sure the inputs are not null
  if ((0 == ncols) || (nullptr == in_groupby_columns) ||
      (nullptr == in_aggregation_column)) {
    return GDF_DATASET_EMPTY;
  }

  // Make sure the output buffers have already been allocated
  if ((nullptr == out_groupby_columns) || (nullptr == out_aggregation_column)) {
    return GDF_DATASET_EMPTY;
  }

  // If there are no rows in the input, return successfully
  if ((0 == in_groupby_columns[0]->size) ||
      (0 == in_aggregation_column->size)) {
    return GDF_SUCCESS;
  }
  
  gdf_error gdf_error_code{GDF_SUCCESS};
  switch (agg_op) {
  case GDF_MIN: {
    gdf_error_code = cudf::type_dispatcher(
        in_aggregation_column->dtype, dispatch_groupby_forwarder<min_op>{},
        ncols, in_groupby_columns, in_aggregation_column, out_groupby_columns,
        out_aggregation_column, ctxt, sorted_indices);
    break;
  }
  case GDF_MAX: {
    gdf_error_code = cudf::type_dispatcher(
        in_aggregation_column->dtype, dispatch_groupby_forwarder<max_op>{},
        ncols, in_groupby_columns, in_aggregation_column, out_groupby_columns,
        out_aggregation_column, ctxt, sorted_indices);
    break;
  }
  case GDF_SUM: {
    gdf_error_code = cudf::type_dispatcher(
        in_aggregation_column->dtype, dispatch_groupby_forwarder<sum_op>{},
        ncols, in_groupby_columns, in_aggregation_column, out_groupby_columns,
        out_aggregation_column, ctxt, sorted_indices);
    break;
  }
  default:
    std::cerr << "Unsupported aggregation method for sort-based groupby."
              << std::endl;
    gdf_error_code = GDF_UNSUPPORTED_METHOD;
  }
  return gdf_error_code;
}