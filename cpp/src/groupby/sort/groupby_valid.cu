/*
 * Copyright 2019 BlazingDB, Inc.
 *     Copyright 2019 Alexander Ocsa <alexander@blazingdb.com>
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "groupby_valid.h"
#include "rmm/thrust_rmm_allocator.h"

#include <hip/hip_runtime.h>

#include "cudf/copying.hpp"
#include "cudf/cudf.h"
#include "utilities/error_utils.hpp"

#include "groupby/aggregation_operations.hpp"

#include "rmm/rmm.h"
#include <limits>
#include <memory>
#include <table/device_table.cuh>
#include <table/device_table_row_operators.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/gather.h>
#include "groupby_valid_helpers.h"

namespace {


template <typename aggregation_type, typename aggregation_operation>
gdf_error GroupbySortWithNulls(
    gdf_size_type num_groupby_cols, int32_t *d_sorted_indices,
    gdf_column *in_groupby_columns[], gdf_column *in_aggregation_column,
    gdf_column *out_groupby_columns[], gdf_column *out_aggregation_column,
    gdf_size_type *out_size, aggregation_operation aggregation_op,
    gdf_context *ctxt) {
  int32_t nrows = in_groupby_columns[0]->size;

  auto device_input_table =
      device_table::create(num_groupby_cols, &(in_groupby_columns[0]));
  auto comp = row_equality_comparator<true>(*device_input_table, true);

  hipStream_t stream;
  hipStreamCreate(&stream);
  auto exec = rmm::exec_policy(stream)->on(stream);
  rmm::device_vector<bool> d_in_agg_col_valids =
      get_bools_from_gdf_valid(in_aggregation_column);

  auto agg_col_iter = thrust::make_permutation_iterator(
      (aggregation_type *)in_aggregation_column->data, d_sorted_indices);
  auto agg_col_valid_iter = thrust::make_permutation_iterator(
      d_in_agg_col_valids.begin(), d_sorted_indices);
  auto agg_col_zip_iter = thrust::make_zip_iterator(
      thrust::make_tuple(agg_col_iter, agg_col_valid_iter));

  rmm::device_vector<bool> d_out_agg_col_valids =
      get_bools_from_gdf_valid(out_aggregation_column);
  auto out_agg_col_zip_iter = thrust::make_zip_iterator(
      thrust::make_tuple((aggregation_type *)out_aggregation_column->data,
                         d_out_agg_col_valids.begin()));

  using op_with_valids = typename aggregation_operation::with_valids;
  op_with_valids agg_op;
  auto ret = thrust::reduce_by_key(exec, d_sorted_indices,
                                   d_sorted_indices + nrows, // input keys
                                   agg_col_zip_iter,         // input values
                                   d_sorted_indices,         // output keys
                                   out_agg_col_zip_iter,     // output values
                                   comp, agg_op);
  auto iter_tuple = ret.second.get_iterator_tuple();

  gdf_size_type new_size =
      thrust::distance((aggregation_type *)out_aggregation_column->data,
                       thrust::get<0>(iter_tuple));

  *out_size = new_size;

  // run gather operation to establish new order
  cudf::table table_in(in_groupby_columns, num_groupby_cols);
  cudf::table table_out(out_groupby_columns, num_groupby_cols);

  cudf::gather(&table_in, d_sorted_indices, &table_out);

  for (int i = 0; i < num_groupby_cols; i++) {
    out_groupby_columns[i]->size = new_size;
  }
  out_aggregation_column->size = new_size;
  set_bools_for_gdf_valid(out_aggregation_column, d_out_agg_col_valids);

  return GDF_SUCCESS;
}

template <typename aggregation_type, template <typename T> class op>
gdf_error typed_groupby_sort(gdf_size_type num_groupby_cols,
                             gdf_column *in_groupby_columns[],
                             gdf_column *in_aggregation_column,
                             gdf_column *out_groupby_columns[],
                             gdf_column *out_aggregation_column,
                             gdf_context *ctxt,
                             rmm::device_vector<int32_t> &sorted_indices) {
  using op_type = op<aggregation_type>;
  gdf_error gdf_error_code = GDF_SUCCESS;
  gdf_size_type output_size{0};

  gdf_error_code = GroupbySortWithNulls<aggregation_type>(
      num_groupby_cols, sorted_indices.data().get(), in_groupby_columns,
      in_aggregation_column, out_groupby_columns, out_aggregation_column,
      &output_size, op_type(), ctxt);
  out_aggregation_column->size = output_size;
  return gdf_error_code;
}

template <template <typename T> class op> struct dispatch_groupby_forwarder {
  template <typename TypeAgg, typename... Ts>
  gdf_error operator()(Ts &&... args) {
    return typed_groupby_sort<TypeAgg, op>(std::forward<Ts>(args)...);
  }
};
} // namespace

gdf_error gdf_group_by_sort_with_valids(
    gdf_size_type ncols, gdf_column *in_groupby_columns[],
    gdf_column *in_aggregation_column, gdf_column *out_groupby_columns[],
    gdf_column *out_aggregation_column, gdf_agg_op agg_op, gdf_context *ctxt,
    rmm::device_vector<int32_t> &sorted_indices) {

  // Make sure the inputs are not null
  if ((0 == ncols) || (nullptr == in_groupby_columns) ||
      (nullptr == in_aggregation_column)) {
    return GDF_DATASET_EMPTY;
  }

  // Make sure the output buffers have already been allocated
  if ((nullptr == out_groupby_columns) || (nullptr == out_aggregation_column)) {
    return GDF_DATASET_EMPTY;
  }

  // If there are no rows in the input, return successfully
  if ((0 == in_groupby_columns[0]->size) ||
      (0 == in_aggregation_column->size)) {
    return GDF_SUCCESS;
  }
  gdf_error gdf_error_code{GDF_SUCCESS};
  switch (agg_op) {
    case GDF_MIN: {
      gdf_error_code = groupby_type_dispatcher(
          in_aggregation_column->dtype, dispatch_groupby_forwarder<min_op>{},
          ncols, in_groupby_columns, in_aggregation_column, out_groupby_columns,
          out_aggregation_column, ctxt, sorted_indices);
      break;
    }
    case GDF_MAX: {
      gdf_error_code = groupby_type_dispatcher(
          in_aggregation_column->dtype, dispatch_groupby_forwarder<max_op>{},
          ncols, in_groupby_columns, in_aggregation_column, out_groupby_columns,
          out_aggregation_column, ctxt, sorted_indices);
      break;
    }
    case GDF_SUM: {
      gdf_error_code = groupby_type_dispatcher(
          in_aggregation_column->dtype, dispatch_groupby_forwarder<sum_op>{},
          ncols, in_groupby_columns, in_aggregation_column, out_groupby_columns,
          out_aggregation_column, ctxt, sorted_indices);
      break;
    }
    default:
      std::cerr << "Unsupported aggregation method for sort-based groupby."
                << std::endl;
      gdf_error_code = GDF_UNSUPPORTED_METHOD;
  }
  return gdf_error_code;
}
