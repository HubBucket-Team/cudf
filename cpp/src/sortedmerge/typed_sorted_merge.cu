#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>
#include <thrust/for_each.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/merge.h>
#include <thrust/sequence.h>

#include "../../src/rmm/thrust_rmm_allocator.h"
#include "../../src/sqls/sqls_rtti_comp.h"

#include "alloc_filtered_cols.cuh"
#include "make_indices.cuh"
#include "pair_rtti.cuh"
#include "soa_info.cuh"
#include "typed_sorted_merge.cuh"

enum side_value { LEFT_SIDE_VALUE = 0, RIGHT_SIDE_VALUE };

gdf_error typed_sorted_merge(gdf_column **     left_cols,
                             gdf_column **     right_cols,
                             const std::size_t ncols,
                             gdf_column *      sort_by_cols,
                             gdf_column *      asc_desc,
                             gdf_column *      output_sides,
                             gdf_column *      output_indices,
                             hipStream_t      cudaStream) {
    GDF_REQUIRE((nullptr != left_cols && nullptr != right_cols),
                GDF_DATASET_EMPTY);

    GDF_REQUIRE(output_sides->dtype == GDF_INT32, GDF_UNSUPPORTED_DTYPE);
    GDF_REQUIRE(output_indices->dtype == GDF_INT32, GDF_UNSUPPORTED_DTYPE);

    const std::size_t left_size  = left_cols[0]->size;
    const std::size_t right_size = right_cols[0]->size;

    const std::size_t total_size = left_size + right_size;
    GDF_REQUIRE(output_sides->size >= total_size, GDF_COLUMN_SIZE_MISMATCH);
    GDF_REQUIRE(output_indices->size >= total_size, GDF_COLUMN_SIZE_MISMATCH);

    GDF_REQUIRE(sort_by_cols->dtype == GDF_INT32, GDF_UNSUPPORTED_DTYPE);
    GDF_REQUIRE(sort_by_cols->size <= ncols, GDF_COLUMN_SIZE_TOO_BIG);

    INITIALIZE_D_VALUES(left);
    INITIALIZE_D_VALUES(right);

    gdf_size_type sort_by_ncols = sort_by_cols->size;

    std::size_t *left_indices  = make_indices(cudaStream, left_size);
    std::size_t *right_indices = make_indices(cudaStream, right_size);

    const thrust::constant_iterator<int> left_side =
        thrust::make_constant_iterator(static_cast<int>(LEFT_SIDE_VALUE));
    const thrust::constant_iterator<int> right_side =
        thrust::make_constant_iterator(static_cast<int>(RIGHT_SIDE_VALUE));

    thrust::zip_iterator<
        thrust::tuple<thrust::constant_iterator<int>, std::size_t *>>
        left_zip_iterator = thrust::make_zip_iterator(
            thrust::make_tuple(left_side, left_indices));
    thrust::zip_iterator<
        thrust::tuple<thrust::constant_iterator<int>, std::size_t *>>
        right_zip_iterator = thrust::make_zip_iterator(
            thrust::make_tuple(right_side, right_indices));

    auto output_zip_iterator = thrust::make_zip_iterator(
        thrust::make_tuple(static_cast<std::size_t *>(output_sides->data),
                           static_cast<std::size_t *>(output_indices->data)));

    void **       filtered_left_d_cols_data  = nullptr;
    void **       filtered_right_d_cols_data = nullptr;
    std::int32_t *filtered_left_d_col_types  = nullptr;
    std::int32_t *filtered_right_d_col_types = nullptr;
    gdf_error     gdf_status = alloc_filtered_d_cols(sort_by_ncols,
                                                 filtered_left_d_cols_data,
                                                 filtered_right_d_cols_data,
                                                 filtered_left_d_col_types,
                                                 filtered_right_d_col_types,
                                                 cudaStream);
    if (GDF_SUCCESS != gdf_status) { return gdf_status; }

    // filter left and right cols for sorting
    std::int32_t *sort_by_d_cols_data =
        reinterpret_cast<std::int32_t *>(sort_by_cols->data);
    thrust::for_each_n(rmm::exec_policy(cudaStream),
                       thrust::make_counting_iterator(0),
                       sort_by_ncols,
                       [=] __device__(const int n) {
                           const std::int32_t n_col = sort_by_d_cols_data[n];

                           void *left_data  = left_d_cols_data[n_col];
                           void *right_data = right_d_cols_data[n_col];

                           filtered_left_d_cols_data[n]  = left_data;
                           filtered_right_d_cols_data[n] = right_data;

                           filtered_left_d_col_types[n] =
                               left_d_col_types[n_col];
                           filtered_right_d_col_types[n] =
                               right_d_col_types[n_col];
                       });

    PairRTTI<std::size_t> comp(
        {
            reinterpret_cast<void **>(filtered_left_d_cols_data),
            filtered_left_d_col_types,
        },
        {
            reinterpret_cast<void **>(filtered_right_d_cols_data),
            filtered_right_d_col_types,
        },
        sort_by_ncols);

    thrust::merge(thrust::device,
                  left_zip_iterator,
                  left_zip_iterator + left_size,
                  right_zip_iterator,
                  right_zip_iterator + right_size,
                  output_zip_iterator,
                  [=] __device__(thrust::tuple<int, std::size_t> left_tuple,
                                 thrust::tuple<int, std::size_t> right_tuple) {
                      const std::size_t left_row  = thrust::get<1>(left_tuple);
                      const std::size_t right_row = thrust::get<1>(right_tuple);

                      return comp.asc_desc_comparison(left_row, right_row);
                  });

    RMM_FREE(left_indices, cudaStream);
    RMM_FREE(right_indices, cudaStream);
    RMM_FREE(filtered_left_d_cols_data, cudaStream);
    RMM_FREE(filtered_right_d_cols_data, cudaStream);
    RMM_FREE(filtered_left_d_col_types, cudaStream);
    RMM_FREE(filtered_right_d_col_types, cudaStream);

    return GDF_SUCCESS;
}
