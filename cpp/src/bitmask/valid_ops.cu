/*
 * Copyright (c) 2018, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
/** ---------------------------------------------------------------------------*
 * @brief Operations on GDF column validity bitmasks
 * 
 * @file column.cpp
 * ---------------------------------------------------------------------------**/
#include <vector>
#include <cassert>

#include "cudf.h"
#include "rmm/rmm.h"
#include "rmm/thrust_rmm_allocator.h"
#include "utilities/error_utils.h"
#include "utilities/cudf_utils.h"
#include "bitmask_ops.h"

#include <thrust/tabulate.h>


/** --------------------------------------------------------------------------*
 * @Synopsis  Counts the number of valid bits for the specified number of rows
 * in the host vector of gdf_valid_type masks
 * 
 * @Param masks The host vector of masks whose bits will be counted
 * @Param num_rows The number of bits to count
 * 
 * @Returns  The number of valid bits in [0, num_rows) in the host vector of masks
 * ----------------------------------------------------------------------------*/
size_t count_valid_bits_host(std::vector<gdf_valid_type> const & masks, int const num_rows)
{
  if((0 == num_rows) || (0 == masks.size())){
    return 0;
  }

  size_t count{0};

  // Count the valid bits for all masks except the last one
  for(size_t i = 0; i < (masks.size() - 1); ++i)
  {
    gdf_valid_type current_mask = masks[i];

    while(current_mask > 0)
    {
      current_mask &= (current_mask-1) ;
      count++;
    }
  }

  // Only count the bits in the last mask that correspond to rows
  int num_rows_last_mask = num_rows % GDF_VALID_BITSIZE;

  if(num_rows_last_mask == 0)
    num_rows_last_mask = GDF_VALID_BITSIZE;

  gdf_valid_type last_mask = *(masks.end() - 1);
  for(int i = 0; (i < num_rows_last_mask) && (last_mask > 0); ++i)
  {
    count += (last_mask & gdf_valid_type(1));
    last_mask >>= 1;
  }

  return count;
}


/* ---------------------------------------------------------------------------*
 * @Synopsis  Counts the number of valid bits for the specified number of rows
 * in a validity bitmask.
 * 
 * @Param[in] masks The validity bitmask buffer in device memory
 * @Param[in] num_rows The number of bits to count
 * @Param[out] count The number of valid bits in the buffer from [0, num_rows)
 * 
 * @Returns  GDF_SUCCESS upon successful completion 
 *
 * ----------------------------------------------------------------------------*/
gdf_error gdf_count_nonzero_mask(gdf_valid_type const * masks, int num_rows, int * count)
{
  gdf_error gdf_status{GDF_SUCCESS};
  
  // TODO: Probably shouldn't create/destroy the stream every time
  hipStream_t count_stream;
  CUDA_TRY(hipStreamCreate(&count_stream));
  
  gdf_status = count_nonzero_mask(masks, num_rows, *count, count_stream);

  CUDA_TRY(hipStreamDestroy(count_stream));

  return gdf_status;
}

/** ---------------------------------------------------------------------------*
 * @brief Concatenate the validity bitmasks of multiple columns
 * 
 * Accounts for the differences between lengths of columns and their bitmasks 
 * (e.g. because gdf_valid_type is larger than one bit).
 * 
 * @param[out] output_mask The concatenated mask
 * @param[in] output_column_length The total length (in data elements) of the 
 *                                 concatenated column
 * @param[in] masks_to_concat The array of device pointers to validity bitmasks
 *                            for the columns to concatenate
 * @param[in] column_lengths An array of lengths of the columns to concatenate
 * @param[in] num_columns The number of columns to concatenate
 * @return gdf_error GDF_SUCCESS or GDF_CUDA_ERROR if there is a runtime CUDA
           error
 * ---------------------------------------------------------------------------**/
gdf_error gdf_mask_concat(gdf_valid_type *output_mask,
                          gdf_size_type output_column_length,            
                          gdf_valid_type *masks_to_concat[], 
                          gdf_size_type *column_lengths, 
                          gdf_size_type num_columns)
{
    // This lambda is executed in a thrust algorithm. Each thread computes and
    // returns one gdf_valid_type element for the concatenated output mask
    auto mask_concatenator = [=] __device__ (gdf_size_type mask_index) {
      gdf_valid_type output_m = 0;
     
      int cur_mask_index = 0, cur_mask_start = 0;
      int cur_mask_len = column_lengths[0];
      
      // Each thread processes one GDF_VALID_BITSIZE worth of valid bits
      for (int bit = 0; bit < GDF_VALID_BITSIZE; ++bit) 
      { 
        gdf_size_type output_index = mask_index * GDF_VALID_BITSIZE + bit;

        // stop when we are beyond the length of the output column (in elements)
        if (output_index >= output_column_length) break;
        
        // find the next column's mask when we step past the current column's length
        while ( (cur_mask_start + cur_mask_len <= output_index) && (cur_mask_index < num_columns - 1) )
        {
          cur_mask_start += cur_mask_len;
          cur_mask_len = column_lengths[++cur_mask_index];           
        }
        
        // Set each valid bit at the right location in this thread's output gdf_valid_type
        // Note: gdf_is_valid returns true when the input mask is a null pointer
        // This makes it behave as if columns with null validity masks have masks of all 1s,
        // which is the desired behavior.
        gdf_size_type index = output_index - cur_mask_start;
        if ( gdf_is_valid(masks_to_concat[cur_mask_index], index) ) 
        {
          output_m |= (1 << bit);     
        }
      }

      return output_m;
    };

    // This is like thrust::for_each where the lambda gets the current index into the output array
    // as input
    thrust::tabulate(rmm::exec_policy(hipStream_t{0}),
                     output_mask,
                     output_mask + gdf_get_num_chars_bitmask(output_column_length),
                     mask_concatenator);

    CUDA_TRY( hipGetLastError() );
        
    return GDF_SUCCESS;
}


